#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>
#include "DEFINE_TYPE.h"
// Texture references for CSR matrix 
texture<int,1> tex_colm;
texture<int2,1> tex_val;


// Scratchpad used by vector dot product for reduction

FLOAT_TYPE* scratchpad;    





// Kernel block and grid parameters - threads in a block and blocks in a grid
#define NUM_THREADS 128
#define NUM_BLOCKS 16

// Macros to simplify kernels 
#define THREAD_ID threadIdx.x+blockIdx.x*blockDim.x
#define THREAD_COUNT gridDim.x*blockDim.x

// Solver parameters - relative tolerance and maximum iterations
#define epsilon 1e-7
#define IMAX 40000

// For timing solver

FLOAT_TYPE utime () {
  struct timeval tv;

  gettimeofday (&tv, NULL);

  return (tv.tv_sec + FLOAT_TYPE (tv.tv_usec) * 1e-6);
}

// Creates a diagonal matrix stored in a vector pcmat, from the CSR matrix findrm, colm, val.
// n is the matrix size.

__global__ void create_jac(int n, int* findrm, int* colm, FLOAT_TYPE* val, FLOAT_TYPE* pcmat)
{
  for(int i=THREAD_ID; i<n; i+=THREAD_COUNT) 
    for(int k=findrm[i]-1; k<findrm[i+1]-1; k++) 
      if(colm[k]==i+1)
        pcmat[i] = 1.0/val[k];
}

// Multiplies diagonal matrix mat stored as a vector by the vector src, storing result in dest.
// n is the vector length.


__global__ void diag_spmv(int n, FLOAT_TYPE *mat, FLOAT_TYPE *src, FLOAT_TYPE *dest) 
{
  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    dest[i] = mat[i]*src[i];
}

// Sets the length-n vector vec to the zero vector.

__global__ void veczero(int n, FLOAT_TYPE* vec) 
{
  for(int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    vec[i] = 0;
}

// Allows fetching FLOAT_TYPE values from texture memory, which only supports integers

static __device__ FLOAT_TYPE fetch_FLOAT_TYPE(texture<int2,1> val, int elem)
{
  int2 v = tex1Dfetch(val, elem);
  return __hiloint2double(v.y, v.x);
}

// Multiplies the CSR matrix in findrm, tex_colm, tex_val by src and stores the
// result in dest. n is the matrix size/vector length.

__global__ void csr_spmv(int n, FLOAT_TYPE* src, FLOAT_TYPE* dest, int *findrm)
{
  for (int row=THREAD_ID; row<n; row+=THREAD_COUNT) {
    dest[row] = 0;
    int a=findrm[row];
    int b=findrm[row+1];
    for (int k=a;k<b;k++)
      dest[row] += fetch_FLOAT_TYPE(tex_val,k-1)*src[tex1Dfetch(tex_colm,k-1)-1];
  }
}

// Computes the dot product of length-n vectors vec1 and vec2. This is reduced in tmp into a
// single value per thread block. The reduced value is stored in the array partial.

__global__ void vecdot_partial(int n, FLOAT_TYPE* vec1, FLOAT_TYPE* vec2, FLOAT_TYPE* partial)
{ 
  __shared__ FLOAT_TYPE tmp[NUM_THREADS];
  tmp[threadIdx.x] = 0;

  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    tmp[threadIdx.x] += vec1[i]*vec2[i];

  for (int i=blockDim.x/2;i>=1;i = i/2) {
    __syncthreads();
    if (threadIdx.x < i) 
      tmp[threadIdx.x] += tmp[i + threadIdx.x]; 
  }

  if (threadIdx.x == 0) 
    partial[blockIdx.x] = tmp[0];
}

// Reduces the output of the vecdot_partial kernel to a single value. The result is stored in result.

__global__ void vecdot_reduce(FLOAT_TYPE* partial, FLOAT_TYPE* result)
{
  __shared__ FLOAT_TYPE tmp[NUM_BLOCKS];

  if (threadIdx.x < NUM_BLOCKS) 
    tmp[threadIdx.x] = partial[threadIdx.x]; 
  else 
    tmp[threadIdx.x] = 0;

  for (int i=blockDim.x/2;i>=1;i = i/2) {
    __syncthreads();
    if (threadIdx.x < i) 
      tmp[threadIdx.x] += tmp[i + threadIdx.x]; 
  }

  if (threadIdx.x == 0) 
    *result = tmp[0];
}

// Divides num by den and stores the result in result. This is very wasteful of the GPU.

__global__ void scalardiv(FLOAT_TYPE* num, FLOAT_TYPE* den, FLOAT_TYPE* result) 
{
  if(threadIdx.x==0 && blockIdx.x==0)
    *result = (*num)/(*den);
}

// Computes r= a*x+y for n-length vectors x and y, and scalar a.

__global__ void axpy(int n, FLOAT_TYPE* a, FLOAT_TYPE* x, FLOAT_TYPE* y, FLOAT_TYPE* r) 
{
  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    r[i] = y[i] + (*a)*x[i];
}

// Computes y= y-a*x for n-length vectors x and y, and scalar a.

__global__ void ymax(int n, FLOAT_TYPE* a, FLOAT_TYPE* x, FLOAT_TYPE* y) 
{
  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    y[i] = y[i] - (*a)*x[i];
}

// Convenient function for performing a vector dot product and reduce all in one go.

void vecdot(int n, FLOAT_TYPE* vec1, FLOAT_TYPE* vec2, FLOAT_TYPE* result) 
{ 
  dim3 BlockDim(NUM_THREADS);
  dim3 GridDim(NUM_BLOCKS);

  vecdot_partial<<<GridDim,BlockDim>>>(n, vec1, vec2, scratchpad);
  vecdot_reduce<<<1,NUM_BLOCKS>>>(scratchpad, result);
}

// Sets dest=src for scalars on the GPU.

void scalarassign(FLOAT_TYPE* dest, FLOAT_TYPE* src)
{
  hipMemcpy(dest, src, sizeof(FLOAT_TYPE), hipMemcpyDeviceToDevice); 
}

// Sets dest=src for n-length vectors on the GPU.

void vecassign(FLOAT_TYPE *dest, FLOAT_TYPE *src, int n) 
{
  hipMemcpy(dest, src, sizeof(FLOAT_TYPE)*n, hipMemcpyDeviceToDevice);
}


// Main solver entry point. Solves Ax = b for x.
// CSR matrix (A) stores the row and column pointer information like Fortran indices (i.e. begin at 1)
// findrm_p     : matrix row pointer
// colm_p       : matrix column pointer
// matrix_val_p : matrix values
// size*        : size of each vector
// b_p          : pointer to RHS vector
// x_p          : solutzion (x) is returned here
extern "C"
void gpucg_solve_(int* findrm_p, int *size_findrm, int* colm_p, int* size_colm, FLOAT_TYPE* matrix_val_p, int *matrix_val_size, 
                             FLOAT_TYPE* b_p, int* rhs_val_size, FLOAT_TYPE *x_p)
{
  // CSR Matrix on the GPU
  int *k_findrm, *k_colm;
  FLOAT_TYPE *k_val;
  // Vectors on the GPU
  FLOAT_TYPE *k_b, *k_x, *k_r, *k_d, *k_q, *k_s;
  // Diagonal matrix on the GPU (stored as a vector)
  FLOAT_TYPE* k_jac;
  // Scalars on the GPU
  FLOAT_TYPE  *k_alpha, *k_snew, *k_beta, *k_sold, *k_s0;

  // Scalars on the host
  FLOAT_TYPE t, s0, snew;
  int iterations = 0;

  // Begin timing
  t = -utime ();

  // Allocate space on the GPU for the CSR matrix and RHS vector, and copy from host to GPU
  hipMalloc((void**)&k_findrm, sizeof(int)*(*size_findrm));
  hipMemcpy(k_findrm, findrm_p, sizeof(int)*(*size_findrm), hipMemcpyHostToDevice);
  hipMalloc((void**)&k_colm, sizeof(int)*(*size_colm));
  hipMemcpy(k_colm, colm_p, sizeof(int)*(*size_colm), hipMemcpyHostToDevice);
  hipBindTexture(NULL, tex_colm, k_colm, sizeof(int)*(*size_colm));
  hipMalloc((void**)&k_val, sizeof(FLOAT_TYPE)*(*matrix_val_size));
  hipMemcpy(k_val, matrix_val_p, sizeof(FLOAT_TYPE)*(*matrix_val_size), hipMemcpyHostToDevice);
  hipMalloc((void**)&k_b, sizeof(FLOAT_TYPE)*(*rhs_val_size));
  hipMemcpy(k_b, b_p, sizeof(FLOAT_TYPE)*(*rhs_val_size), hipMemcpyHostToDevice);

  // Allocate space for vectors on the GPU
  hipMalloc((void**)&k_x, sizeof(FLOAT_TYPE)*(*rhs_val_size));
  hipMalloc((void**)&k_r, sizeof(FLOAT_TYPE)*(*rhs_val_size));
  hipMalloc((void**)&k_d, sizeof(FLOAT_TYPE)*(*rhs_val_size));
  hipMalloc((void**)&k_q, sizeof(FLOAT_TYPE)*(*rhs_val_size));
  hipMalloc((void**)&k_s, sizeof(FLOAT_TYPE)*(*rhs_val_size));
  hipMalloc((void**)&k_jac, sizeof(FLOAT_TYPE)*(*rhs_val_size));
  hipMalloc((void**)&k_alpha, sizeof(FLOAT_TYPE));
  hipMalloc((void**)&scratchpad, sizeof(FLOAT_TYPE)*NUM_BLOCKS);
  hipMalloc((void**)&k_snew, sizeof(FLOAT_TYPE)*NUM_BLOCKS);
  hipMalloc((void**)&k_sold, sizeof(FLOAT_TYPE));
  hipMalloc((void**)&k_beta, sizeof(FLOAT_TYPE));
  hipMalloc((void**)&k_s0, sizeof(FLOAT_TYPE));

  // Dimensions of blocks and grid on the GPU
  dim3 BlockDim(NUM_THREADS);
  dim3 GridDim(NUM_BLOCKS);

  struct timeval t1, t2;    
  gettimeofday(&t1, 0);

  // Create diagonal preconditioning matrix (J = 1/diag(M)) 
  create_jac<<<1,BlockDim>>>(*rhs_val_size, k_findrm, k_colm, k_val, k_jac);

  // Bind the matrix to the texture cache - this was not done earlier as we modified the matrix
  hipBindTexture(NULL, tex_val, k_val, sizeof(FLOAT_TYPE)*(*matrix_val_size)); 

  // Initialise result vector (x=0)
  veczero<<<1,BlockDim>>>(*rhs_val_size, k_x);

  // r=b-Ax (r=b since x=0), and d=M^(-1)r
  hipMemcpy(k_r, k_b, sizeof(FLOAT_TYPE)*(*rhs_val_size), hipMemcpyDeviceToDevice);
  diag_spmv<<<1,BlockDim>>>(*rhs_val_size, k_jac, k_r, k_d);

  // s0 = r.d
  vecdot(*rhs_val_size, k_r, k_d, k_s0);
  // snew = s0
  scalarassign(k_snew, k_s0);

  // Copy snew and s0 back to host so that host can evaluate stopping condition
  hipMemcpy(&snew, k_snew, sizeof(FLOAT_TYPE), hipMemcpyDeviceToHost);
  hipMemcpy(&s0, k_s0, sizeof(FLOAT_TYPE), hipMemcpyDeviceToHost);

  // While i < imax and snew > epsilon^2*s0
  while (iterations < IMAX && snew > epsilon*epsilon*s0)
  {
    // q = Ad
    csr_spmv<<<GridDim,BlockDim>>>(*rhs_val_size, k_d, k_q, k_findrm);
    // alpha = snew/(d.q)
    vecdot(*rhs_val_size, k_d, k_q, k_alpha);
    scalardiv<<<1,1>>>(k_snew, k_alpha, k_alpha);
    // x = x + alpha*d
    axpy<<<GridDim,BlockDim>>>(*rhs_val_size, k_alpha, k_d, k_x, k_x);
    // r = r - alpha*q
    ymax<<<GridDim,BlockDim>>>(*rhs_val_size, k_alpha, k_q, k_r);
    // s = M^(-1)r
    diag_spmv<<<GridDim,BlockDim>>>(*rhs_val_size, k_jac, k_r, k_s);
    // sold = snew
    scalarassign(k_sold, k_snew);
    // snew = r.s
    vecdot(*rhs_val_size, k_r, k_s, k_snew);
    // beta = snew/sold
    scalardiv<<<1,1>>>(k_snew, k_sold, k_beta);
    // d = s + beta*d
    axpy<<<GridDim,BlockDim>>>(*rhs_val_size, k_beta, k_d, k_s, k_d);
    // Copy back snew so the host can evaluate the stopping condition
    hipMemcpy(&snew, k_snew, sizeof(FLOAT_TYPE), hipMemcpyDeviceToHost);
    // i = i+1
    iterations++;
  }

  gettimeofday(&t2, 0);
  FLOAT_TYPE time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

  printf("Time of GPU:  %3.1f ms \n", time);

  // Copy result vector back from GPU
  hipMemcpy(x_p, k_x, sizeof(FLOAT_TYPE)*(*rhs_val_size), hipMemcpyDeviceToHost);

  // Clean up
  hipUnbindTexture(tex_colm);
  hipUnbindTexture(tex_val);
  hipFree(k_findrm);
  hipFree(k_colm);
  hipFree(k_val);
  hipFree(k_b);
  hipFree(k_x);
  hipFree(k_r);
  hipFree(k_d);
  hipFree(k_q);
  hipFree(k_jac);
  hipFree(k_alpha);
  hipFree(k_snew);
  hipFree(k_sold);
  hipFree(k_beta);
  hipFree(k_s0);
  hipFree(scratchpad);

  // End timing - call hipDeviceSynchronize so we know all computation is finished before we stop the clock.
  hipDeviceSynchronize();
  t += utime ();

  // Interesting information
  printf("Iterations: %d \n", iterations);
  printf("CUDA error is: %s \n", hipGetErrorString(hipGetLastError()));
  printf("Solve time: %f seconds\n", t);
}

