#include "hip/hip_runtime.h"
#include "pcg_gpu.h"
#include <iostream>
#include "hip/hip_runtime.h"
// #include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include ""
#include <fstream>
#include <assert.h>
#include <cmath>
#include <vector>
#include "hipblas.h"

// #include <Eigen/Dense>

#define N_MAX 2000
#define NNZ_MAX 13000

__constant__ int IA[N_MAX+1];
__constant__ int JA[NNZ_MAX];

void readIAandJA(const int size_Matrix,const int size_nozeronumber,const int *IAtemp, const int *JAtemp)
{
	hipMalloc((void**)&IA, sizeof(int)*(size_Matrix + 1));
	hipMalloc((void**)&JA, sizeof(int)*size_nozeronumber);
	hipMemcpyToSymbol(HIP_SYMBOL(IA), IAtemp, sizeof(int)*(size_Matrix + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(JA), JAtemp, sizeof(int)*size_nozeronumber);
}

template<typename DOUBLE>
__global__ void initialvalue(int N, DOUBLE *A, DOUBLE *B, DOUBLE *Minverse, DOUBLE *r, DOUBLE *z, DOUBLE *p)
{
	int blockId = blockIdx.y*gridDim.x + blockIdx.x;
	int tid = blockId * (blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;
	while (tid < N)
	{
		int jtmp = IA[tid + 1] - IA[tid];
		for (int j = 0; j < jtmp; j++)
		{
			if (JA[j + IA[tid]] == tid)
			{
				Minverse[tid] = 1.0 / A[j + IA[tid]];
			}
		}
		r[tid] = B[tid];
		z[tid] = Minverse[tid] * r[tid];
		p[tid] = z[tid];
		tid += (gridDim.x*blockDim.x)*(gridDim.y*blockDim.y);
	}
}

template<typename DOUBLE>
__global__ void VectorAMUtiplyP(int N, DOUBLE *A, DOUBLE *p, DOUBLE *ap)
{
	int blockId = blockIdx.y*gridDim.x + blockIdx.x;
	int tid = blockId * (blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;
	while (tid < N)
	{
		DOUBLE temp = 0;
		int jtemp;
		jtemp = IA[tid + 1] - IA[tid];
		for (int j = 0; j < jtemp; j++)
		{
			temp += A[j + IA[tid]] * p[JA[j + IA[tid]]];
		}
		ap[tid] = temp;
		tid += (gridDim.x*blockDim.x)*(gridDim.y*blockDim.y);
	}
}

template<typename DOUBLE>
__global__ void inerate_ak(DOUBLE *zr, DOUBLE *pap, DOUBLE *ak)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		*ak = (*zr) / (*pap);
	}
}

template<typename DOUBLE>
__global__ void inerate_x(int N, DOUBLE *p, DOUBLE *ak, DOUBLE *x)
{
	int blockId = blockIdx.y*gridDim.x + blockIdx.x;
	int tid = blockId * (blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;
	while (tid < N)
	{
		x[tid] = x[tid] + (*ak) * p[tid];
		tid += (gridDim.x*blockDim.x)*(gridDim.y*blockDim.y);
	}
}

template<typename DOUBLE>
__global__ void inerate_r(int N, DOUBLE *ak, DOUBLE *ap, DOUBLE *r)
{
	int blockId = blockIdx.y*gridDim.x + blockIdx.x;
	int tid = blockId * (blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;
	while (tid < N)
	{
		r[tid] = r[tid] - (*ak)*ap[tid];
		tid += (gridDim.x*blockDim.x)*(gridDim.y*blockDim.y);
	}
}

template<typename DOUBLE>
__global__ void inerate_z(int N, DOUBLE *Minverse, DOUBLE *r, DOUBLE *z)
{
	int blockId = blockIdx.y*gridDim.x + blockIdx.x;
	int tid = blockId * (blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;
	while (tid < N)
	{
		z[tid] = Minverse[tid] * r[tid];
		tid += (gridDim.x*blockDim.x)*(gridDim.y*blockDim.y);
	}
}

template<typename DOUBLE>
__global__ void inerate_p(int N, DOUBLE *zrnew, DOUBLE *zr, DOUBLE *z, DOUBLE *p)
{
	int blockId = blockIdx.y*gridDim.x + blockIdx.x;
	int tid = blockId * (blockDim.x *blockDim.y) + (threadIdx.y*blockDim.x) + threadIdx.x;
	while (tid < N)
	{
		p[tid] = z[tid] + ((*zrnew) / (*zr))*p[tid];
		tid += (gridDim.x*blockDim.x)*(gridDim.y*blockDim.y);
	}
}

template<typename DOUBLE>
void PCG<DOUBLE>::function_pcg(const int Ntemp, const int NNZtemp, const DOUBLE *Atemp, const DOUBLE *Btemp, DOUBLE* x)
{

        DOUBLE *A;
        DOUBLE *B;
        int N;
	N = Ntemp;
	assert(N <= N_MAX);
	assert(NNZtemp <= NNZ_MAX);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	dim3 block(32, 32);
	dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
	DOUBLE *dev_Minverse, *dev_r, *dev_z, *dev_p;
	DOUBLE *zr = new DOUBLE, *dev_zr;
	DOUBLE *dev_ap;
	DOUBLE *pap = new DOUBLE, *dev_pap;
	DOUBLE *ak = new DOUBLE, *dev_ak;
	DOUBLE *dev_x;
	DOUBLE *zrnew = new DOUBLE, *dev_zrnew;

	hipMalloc((void**)&A, sizeof(DOUBLE)*NNZtemp);
	hipMalloc((void**)&B, sizeof(DOUBLE)*N);
	hipMalloc((void**)&dev_Minverse, sizeof(DOUBLE)*N);
	hipMalloc((void**)&dev_r, sizeof(DOUBLE)*N);
	hipMalloc((void**)&dev_z, sizeof(DOUBLE)*N);
	hipMalloc((void**)&dev_p, sizeof(DOUBLE)*N);
	hipMalloc((void**)&dev_zr, sizeof(DOUBLE));
	hipMalloc((void**)&dev_ap, sizeof(DOUBLE)*N);
	hipMalloc((void**)&dev_pap, sizeof(DOUBLE));
	hipMalloc((void**)&dev_ak, sizeof(DOUBLE));
	hipMalloc((void**)&dev_x, sizeof(DOUBLE)*N);
	hipMalloc((void**)&dev_zrnew, sizeof(DOUBLE));
	
	hipMemcpy(A, Atemp, sizeof(DOUBLE)*NNZtemp, hipMemcpyHostToDevice);
	hipMemcpy(B, Btemp, sizeof(DOUBLE)*N, hipMemcpyHostToDevice);


	initialvalue << <grid, block >> > (N, A, B, dev_Minverse, dev_r, dev_z, dev_p);
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		std::cout << "CUBLAS����ʵ��������" << std::endl;
		getchar();
	}
	for (int i = 0; i < N; i++)
	{
		hipblasDdot(handle, N, dev_z, 1, dev_r, 1, dev_zr);
		VectorAMUtiplyP << <grid, block >> > (N, A, dev_p, dev_ap);
		hipblasDdot(handle, N, dev_ap, 1, dev_p, 1, dev_pap);
		inerate_ak << <grid, block >> > (dev_zr, dev_pap, dev_ak);
		inerate_x << <grid, block >> > (N, dev_p, dev_ak, dev_x);
		inerate_r << <grid, block >> > (N, dev_ak, dev_ap, dev_r);
		inerate_z << <grid, block >> > (N, dev_Minverse, dev_r, dev_z);
		hipblasDdot(handle, N, dev_z, 1, dev_r, 1, dev_zrnew);
		hipMemcpy(zrnew, dev_zrnew, sizeof(DOUBLE), hipMemcpyDeviceToHost);
		if (sqrt(*zrnew) < 1.0e-8) break;
		inerate_p << <grid, block >> > (N, dev_zrnew, dev_zr, dev_z, dev_p);
	}
	hipMemcpy(x, dev_x, sizeof(DOUBLE)*N, hipMemcpyDeviceToHost);
        


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	std::cout << time;
	hipFree(A);
	hipFree(B);
	hipFree(dev_Minverse);
	hipFree(dev_r);
	hipFree(dev_z);
	hipFree(dev_p);
	hipFree(dev_zr);
	hipFree(dev_ap);
	hipFree(dev_pap);
	hipFree(dev_ak);
	hipFree(dev_x);
	hipFree(dev_zrnew);
}

// template __global__ void initialvalue<double>;
// template __global__ void initialvalue<float>;
// template __global__ void VectorAMUtiplyP<double>;
// template __global__ void VectorAMUtiplyP<float>;
// template __global__ void inerate_ak<double>;
// template __global__ void inerate_ak<float>;
// template __global__ void inerate_x<double>;
// template __global__ void inerate_x<float>;
// template __global__ void inerate_r<double>;
// template __global__ void inerate_r<float>;
// template __global__ void inerate_z<double>;
// template __global__ void inerate_z<float>;
// template __global__ void inerate_p<double>;
// template __global__ void inerate_p<float>;

